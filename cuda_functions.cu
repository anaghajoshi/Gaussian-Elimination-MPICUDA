#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_functions.h"
/*This is for the CUDA only code implemented by using the exact algorithm described by the book*/

#define BLOCK_SIZE 8

float *d_A;    //device matrix
float *d_x;    //device array
int *d_loc;    //device array
float *d_y;   //device array

dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
dim3 grid(1,1);	

//Reads matrix from a file
void read_matrix(char* filename, int *n, float **B_1)
{
  float **B;
  int i,test;
  int n2 = (*n)+1;

  FILE *fh=fopen(filename,"rb");
  if(fh != NULL)
  {
    test=fread(n,sizeof(int),1,fh);
    (*B_1) = (float *)malloc((*n)*n2*sizeof(float));
    B = (float **)malloc((*n)*sizeof(float *));

    for(i=0; i<(*n); i++) 
      B[i] = &(*B_1)[i*n2];

    fseek(fh,4,SEEK_SET);
      test = fread((*B_1), sizeof(float), (*n)*n2, fh);
  }

  if(test!=1)
  {
    test=1;
    fflush(stdout);
  }
  fclose(fh);
}

//Prints matrix
void print_matrix(int n, float *B)
{
  int i,j;
  for (i=0; i<n; i++)
  {
    for(j=0; j<n+1; j++) 
      printf("%f\t",B[i*(n+1)+j]);
    printf("\n");
  }
  printf("\n");
}

__global__ void print(float* A, int N)
{
  int iter = threadIdx.x;

  if(iter == 1)
  {
    int i,j;
    for (i=0; i<N; i++)
    {
      for(j=0; j<N+1; j++) 
        printf("%f\t",A[i*(N+1)+j]);
      printf("\n");
    }
  }
  printf("\n");
}

__global__ void find_max(float* A, int* loc, int N, int row_bound)
{
  //find pivot row picked
  float magnitude = 0;
  int picked, j;

  int iter = threadIdx.x;

  if(iter == 1)
  {
    for(j = row_bound; j < N; j++)
    {
      if( abs(A[loc[j]*(N+1) + row_bound]) > magnitude)
      {
        magnitude = abs(A[loc[j]*(N+1) + row_bound]);
        picked = j;
      }
    }
    int temp = loc[row_bound];
    loc[row_bound] = loc[picked];
    loc[picked] = temp;
  }
}

__global__ void con_up(float* A, int N, int row_bound)
{
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  if( (row > row_bound) && (col >= row_bound) )
  {
    float t = A[row*(N+1) + row_bound]/A[row_bound*(N+1) + row_bound];
    //printf("Num:%f Den:%f\n", A[row*(N+1) + row_bound], A[row_bound*(N+1) + row_bound]);
    //printf("\n");

    //if(row == (row_bound+1))
    //  printf("%f\n", t);

    A[row*(N+1) + col] = A[row*(N+1) + col] - A[row_bound*(N+1) + col]*t;
    
if(col == N-1)
      A[row*(N+1) + (col+1)] = A[row*(N+1) + (col+1)] - A[row_bound*(N+1) + (col+1)]*t;
    A[row*(N+1)+row_bound] = t;
  }
}

__global__ void reduce1(float* A, float* x, float* y, int N, int row_bound)
{
  y[row_bound] = A[row_bound*(N+1) + N];
int row = blockIdx.y*blockDim.y + threadIdx.y;
if(row < row_bound)
    A[row*(N+1) + N] = A[row*(N+1) + N] - y[row_bound]*A[row*(N+1) + row_bound];
 
}

__global__ void reduce2(float* A, float* x, float* y, int N, int row_bound)
{
  x[row_bound] = y[row_bound]/A[row_bound*(N+1) + row_bound];
  //printf("%f\n", x[row_bound]);

  int row = blockIdx.y*blockDim.y + threadIdx.y;
  //printf("%d\n", row);

  if(row < row_bound)
    y[row] = y[row] - x[row_bound]*A[row*(N+1) + row_bound];
}


__global__ void printvf(float* A, int N)
{
  int iter = threadIdx.x;

  if(iter == 1)
  {
    int i;
    for (i=0; i<N; i++)
      printf("%f\n",A[i]);
  }
}

__global__ void printv(int* A, int N)
{
  int iter = threadIdx.x;

  if(iter == 1)
  {
    int i;
    for (i=0; i<N; i++)
      printf("%d\n",A[i]);
  }
  printf("\n");
}

void gpuAlloc(int x, int y)
{
  hipMalloc((void **)&d_A, x*sizeof(float));
  hipMalloc((void **)&d_x, y*sizeof(float));
  hipMalloc((void **)&d_y, y*sizeof(float));
  hipMalloc((void **)&d_loc, y*sizeof(int));
  dim3 grid(y/BLOCK_SIZE, y/BLOCK_SIZE);	
}

void gpuFree(void)
{
  hipFree(d_A);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_loc);
}

void gpuMemLoad(int x, int y, float** h_A, float** h_x, int** h_loc)
{
  hipMemcpy(d_A, *h_A, x*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_x, *h_x, y*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_loc, *h_loc, y*sizeof(int), hipMemcpyHostToDevice);
}

void gpuMemStore(int x, int y, float** h_A, float** h_x, float** h_y)
{
  hipMemcpy(*h_A, d_A, x*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(*h_x, d_x, y*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(*h_y, d_y, y*sizeof(float), hipMemcpyDeviceToHost);
}

void gpuLoopStore(int x, float** h_A)
{
  hipMemcpy(*h_A, d_A, x*sizeof(float), hipMemcpyDeviceToHost);
}

void gpuFindMax(int x, int i)
{
  find_max<<<1,2>>>(d_A, d_loc, x, i);
}

void gpuConUp(int x, int i)
{
  con_up<<<grid, threadBlock>>>(d_A, x, i);
}

void gpuBackSubs(int x, int i)
{
  reduce2<<<grid, threadBlock>>>(d_A, d_x, d_y,x, i);
}
void gpuFwdSubs(int x, int i)
{
  reduce1<<<grid, threadBlock>>>(d_A, d_x, d_y, x, i);
}

