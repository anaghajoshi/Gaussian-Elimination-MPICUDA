#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_functions2.h"
/*This is for the hybrid implementation*/

#define BLOCK_SIZE 8

float *d_A;    //device matrix
float *d_pivot; //device array
int *d_loc;    //device array
float *d_x;    //device array
float *d_y;   //device array

dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
dim3 grid(1,1);	

__global__ void con_up(float* A, float* d_pivot, int* loc, int N, int rank, int size, int row_bound)
{
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  int col_bound = rank + row*size;  

  if( (row < N/size) && (col >= row_bound) && (col_bound > row_bound))
  {
    float t = A[row*(N+1) + row_bound]/d_pivot[0];

    A[row*(N+1) + col] = A[row*(N+1) + col] - d_pivot[col - row_bound]*t;
    if(col == N-1)
      A[row*(N+1) + (col+1)] = A[row*(N+1) + (col+1)] - d_pivot[(col+1) - row_bound]*t;
    A[row*(N+1)+row_bound] = t;
 }
}

__global__ void reduce1(float* A, int N, int row_bound, int rank, int size,float y)
{
  int row = blockIdx.y*blockDim.y + threadIdx.y;

  int a = rank+row*size;

  if(a < row_bound)
    A[row*(N+1) + N] = A[row*(N+1) + N] - y*A[row*(N+1) + row_bound];
  
}

__global__ void reduce2(float* A, int N, int row_bound, int rank, int size, float x, float* y)
{
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int a =rank + row*size;

  if(row < row_bound)
    y[a] = y[a] - x*A[row*(N+1) + row_bound];
}


void gpuAlloc(int x, int y)
{
  hipMalloc((void **)&d_A, x*sizeof(float));
  hipMalloc((void **)&d_loc, y*sizeof(int));
  hipMalloc((void **)&d_x, y*sizeof(float));
  hipMalloc((void **)&d_y, y*sizeof(float));
  dim3 grid(y/BLOCK_SIZE, y/BLOCK_SIZE);	
}

void gpuFree(void)
{
  hipFree(d_A);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_loc);
  hipFree(d_pivot);
}

void gpuLoopAlloc(int y)
{
  hipMalloc((void **)&d_pivot, y*sizeof(float));
}

void gpuMemLoad(int x, int y, float** h_A, float** h_x, int** h_loc)
{
  hipMemcpy(d_A, *h_A, x*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_loc, *h_loc, y*sizeof(int), hipMemcpyHostToDevice);
}

void gpuMemStore(int x, int y, float** h_A, float** h_x, int** h_loc, float** h_y)
{
  hipMemcpy(*h_A, d_A, x*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(*h_loc, d_loc, y*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*h_x, d_x, y*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(*h_y, d_y, y*sizeof(float), hipMemcpyDeviceToHost);
}

void gpuLoopLoad(int x, int y, float** pivot, int ** h_loc)
{
  hipMemcpy(d_pivot, *pivot, x*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_loc, *h_loc, y*sizeof(int), hipMemcpyHostToDevice);
}

void gpuLoopStore(int x, float** h_A)
{
  hipMemcpy(*h_A, d_A, x*sizeof(float), hipMemcpyDeviceToHost);
}

void gpuLoopStore2(int x, float** y)
{
  hipMemcpy(*y, d_y, x*sizeof(float), hipMemcpyDeviceToHost);
}

void gpuConUp(int i, int num, int rank, int size)
{
  con_up<<<grid, threadBlock>>>(d_A, d_pivot, d_loc, num, rank, size, i);
}

void gpuBackSubs(int i, int x, int rank, int size, float temp_result, float** y)
{
  hipMemcpy(d_y, *y, x*sizeof(float), hipMemcpyHostToDevice);
  reduce2<<<grid, threadBlock>>>(d_A, x, i, rank, size, temp_result, d_y);
}

void gpuFwdSubs(int i, int x, int rank, int size, float temp_result)
{
  reduce1<<<grid, threadBlock>>>(d_A, x, i, rank, size, temp_result);
}

