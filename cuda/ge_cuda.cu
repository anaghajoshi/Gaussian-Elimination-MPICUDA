#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_functions.h"

int main(int argc, char *argv[])
{
  int i;
  int num, matSize;
  int ret, opt;

  char *ifile = (char*)malloc(30);
  strncpy(ifile, "default_file.dat", 30);
  
  while((opt=getopt(argc,argv,"i:"))!=-1)
  {
    switch(opt)
    {
      case 'i':
	ifile=strdup(optarg);
	break;
    }
  }

  hipEvent_t start, stop, exec_start, exec_stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&exec_start);
  hipEventCreate(&exec_stop);

  hipEventRecord(start);

  FILE *fh = fopen(ifile, "rb");
  if(fh != NULL)
  {
    ret = fread(&num, sizeof(int), 1, fh);
    if(ret != 1)
      printf("Improper read operation");
    fclose(fh);
  }

  float *h_A;  //host matrix

  int *h_loc = (int *) malloc (sizeof(int)*num);
  for(i = 0; i < num; i++)
    h_loc[i] = i;

  float *h_x = (float *) malloc (sizeof(float)*num);
  for(i = 0; i < num; i++)
    h_x[i] = 0;
  
  float *h_y = (float *) malloc (sizeof(float)*num);
  for(i = 0; i < num; i++)
    h_y[i] = 0;
  
  matSize = num*(num+1);

  read_matrix(ifile, &num, &h_A);

  hipEventRecord(start);

  //Allocating memory in device
  gpuAlloc(matSize, num);

  //Copy Data to Device
  gpuMemLoad(matSize, num, &h_A, &h_x, &h_loc);

  for(i = 0; i < num; i++)
  {
    //Kernel Call 1
    gpuConUp(num, i);
  }

 for(i = 0; i < num; i++)
  {
    //Kernel Call 2
    gpuFwdSubs(num, i);
  }

  for(i = num-1; i > -1; i--)
  {
    //Kernel Call 3
    gpuBackSubs(num, i);
  }

  gpuMemStore(matSize, num, &h_A, &h_x, &h_y);

  hipEventRecord(stop);
  hipEventRecord(exec_stop);
//  printf("LU is:\n");
//  print_matrix(num,h_A);

/*  
printf("Result Y  is:\n");
  for(i = 0; i < num; i++)
    printf("y[%d]:%f\n", i, h_y[i]);

  printf("Result array is:\n");
  for(i = 0; i < num; i++)
    printf("x[%d]:%f\n", i, h_x[i]);
*/
  hipEventSynchronize(stop);
  hipEventSynchronize(exec_stop);
  float milliseconds = 0;
  float execmilliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventElapsedTime(&execmilliseconds, exec_start, exec_stop);
  printf("Time:%f\n", milliseconds);
  printf("Computation Time:%f\n", execmilliseconds);

  free(h_A);
  free(h_loc);
  free(h_x);

  //Free Device memory
  gpuFree();

  return 0;
}
